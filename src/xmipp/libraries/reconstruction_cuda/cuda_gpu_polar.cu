#include "hip/hip_runtime.h"
/***************************************************************************
 *
 * Authors:    David Strelak (davidstrelak@gmail.com)
 *
 * Unidad de  Bioinformatica of Centro Nacional de Biotecnologia , CSIC
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
 * 02111-1307  USA
 *
 *  All comments concerning this program package may be sent to the
 *  e-mail address 'xmipp@cnb.csic.es'
 ***************************************************************************/

#include "cuda_basic_math.h"
#include "core/xmipp_macros.h"

template<typename T, bool FULL_CIRCLE>
__global__
void polarFromCartesian(const T *__restrict__ in, int inX, int inY,
        T *__restrict__ out, int samples, int rings, int signals, int posOfFirstRing)
{
    // input is 2D signal - each row is a ring of samples
    // map thread to sample in the polar coordinate
    int s = (blockIdx.x*blockDim.x + threadIdx.x) % samples; // sample position == column
    int n = (blockIdx.x*blockDim.x + threadIdx.x) / samples; // signal index

    if ((n >= signals) || (s >= samples)) return;

    T piConst = FULL_CIRCLE ? 2 * M_PI : M_PI;
    T dphi = piConst / (T)samples;
    T phi = s * dphi;

    T sinPhi = sin(phi);
    T cosPhi = cos(phi);

    // transform current polar position to cartesian
    // shift origin to center of the input image
    for (int r = 0; r < rings; ++r) {
        T cartX = sinPhi * (T)(r + posOfFirstRing) + (int)(inX / (T)2);
        T cartY = cosPhi * (T)(r + posOfFirstRing) + (int)(inY / (T)2);

        int offset = (n * samples * rings) + (r * samples) + s;
        // Bilinear interpolation
        // we don't wrap, as we expect that the biggest ring has some edge around, so we cannot read
        // data out of domain
        T val = biLerp(in + (n * inX * inY),
                inX, inY,
                cartX , cartY);
//        printf("sample: [%d %d+%d=%d %d] reading from [%f %f] value %f (stored at %d)\n",
//                s, r,firstRing, r + firstRing, n,
//                cartX, cartY,
//                val, offset);

        // Nearest neighbour interpolation
//        int cartXRound = (int)(cartX + (T)0.5) - FIRST_XMIPP_INDEX(inX);
//        int cartYRound = (int)(cartY + (T)0.5) - FIRST_XMIPP_INDEX(inY);
//        T val = in[(n * inX * inY) + (cartYRound * inX) + cartXRound];
//        printf("sample: [%d %d+%d=%d %d] reading from [%f %f] value %f (stored at %d)\n",
//                s, r,firstRing, r + firstRing, n,
//                cartXRound, cartYRound,
//                val, offset);
        out[offset] = val;
    }
}
