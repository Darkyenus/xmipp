#include "hip/hip_runtime.h"
#include "core/xmipp_macros.h"
#include "assert.h"

template<typename T, int degree, bool wrap>
__global__
void applyGeometryKernel_2D_wrap(const T* trInv, T minxpp, T maxxpp, T minypp,
        T maxypp, T minxp, T maxxp, T minyp, T maxyp, T* data, int xdim,
        int ydim, T* coefs, int coefsXDim, int coefsYDim) {
    // assign output pixel to thread
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= xdim || i >= ydim)
        return;

    // Calculate this position in the input image according to the
    // geometrical transformation
    // they are related by
    // coords_output(=x,y) = A * coords_input (=xp,yp)
    T xp = j * trInv[0] + i * trInv[1] + trInv[2];
    T yp = j * trInv[3] + i * trInv[4] + trInv[5];

    if (wrap) {
        bool x_isOut = XMIPP_RANGE_OUTSIDE_FAST(xp, minxpp, maxxpp);
        bool y_isOut = XMIPP_RANGE_OUTSIDE_FAST(yp, minypp, maxypp);

        if (x_isOut) {
            xp = realWRAP(xp, minxp - 0.5, maxxp + 0.5); // FIXME specialize for float/double
        }

        if (y_isOut) {
            yp = realWRAP(yp, minyp - 0.5, maxyp + 0.5);
        }

        switch (degree) {
        case 0:
        case 1:
        case 2:
            assert("degree 0..2 not implemented");
            break;
        case 3: {
			T res = interpolatedElementBSpline2D_Degree3(xp, yp, coefsXDim,
                    coefsYDim, coefs);
            size_t index = i * xdim + j;
            data[index] = res;
        }
            break;
        default:
            printf("Degree %d is not supported\n", degree);
        }
    } else {
        assert("non-wrap not implemented");
    }
}

template<typename T, int degree>
__global__
void applyLocalShiftGeometryKernel(const T* coefsX, const T *coefsY,
	T* output, int xdim, int ydim, int ndim,
	T* input, int curFrame,
	int lX, int lY, int lN) { // number of control points in each dim
    // assign output pixel to thread
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= xdim || y >= ydim)
        return;

    // Calculate this position in the input image according to the
    // geometrical transformation
    
    T delta = 0.0001;
    T hX = xdim / (T)(lX-1);
    T hY = ydim / (T)(lY-1);
    T hT = ndim / (T)(lN-1);
    
	T shiftX = 0;
	T shiftY = 0;
	// compute influence of each control point
	for (int j = 0; j < (lN+2)*(lY+2)*(lX+2); ++j) {
	    int controlIdxT = j/((lY+2)*(lX+2))-1;
	    int XY=j%((lX+2)*(lY+2));
	    int controlIdxY = (XY/(lX+2)) -1;
	    int controlIdxX = (XY%(lX+2)) -1;
	    // note: if control point is not in the tile vicinity, val == 0 and can be skipped
	    T tmp = bspline03((x / (T)hX) - controlIdxX) *
	            bspline03((y / (T)hY) - controlIdxY) *
	            bspline03((curFrame / (T)hT) - controlIdxT);
	    if (fabsf(tmp) > delta) {
	        size_t coeffOffset = (controlIdxT+1) * (lX+2)*(lY+2) + (controlIdxY+1) * (lX+2) + (controlIdxX+1);
	        shiftX += coefsX[coeffOffset] * tmp;
	        shiftY += coefsY[coeffOffset] * tmp;
	    }
	}
	
	switch (degree) {
        case 0:
        case 1:
        case 2:
            assert("degree 0..2 not implemented");
            break;
        case 3: {
			T res = interpolatedElementBSpline2D_Degree3(x - shiftX, y - shiftY, xdim,
			                    ydim, input);
		    size_t index = y * xdim + x;
		    output[index] = res;
	    }
            break;
        default:
            printf("Degree %d is not supported\n", degree);
        }
}

